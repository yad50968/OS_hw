
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<inttypes.h>

#define PAGESIZE 32
#define PHYSICAL_MEM_SIZE 32768
#define STORAGE_SIZE 131072
#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"
typedef unsigned char uchar;
typedef uint32_t u32;
__device__ __managed__ int t=0;
__device__ __managed__ int PAGE_ENTRIES = 0;
__device__ __managed__ int PAGEFAULT = 0;
__device__ __managed__ uchar storage[STORAGE_SIZE];
__device__ __managed__ uchar results[STORAGE_SIZE];
__device__ __managed__ uchar input[STORAGE_SIZE];
__device__ __managed__ u32 counter[1024];

extern __shared__ u32 pt[];

__device__ u32 paging(uchar *buffer,u32 addr, uchar value)
{
	t = t+1;
	int i;

	for(i=0;i<1024;i++){
		if(pt[i]==addr){
			counter[i] = t;			
			return i*32+value; // hit
		}
	}
	for(i=0;i<1024;i++){
		if(pt[i]== -1){
				PAGEFAULT++;				
				counter[i] = t;
				pt[i]=addr;
				return i*32+value;
		}
	}
	
	    	
		int swap_index=0; 
		int min=counter[0];
		for(int j=0;j<1024;j++){
				if(counter[j]<min){
					min = counter[j];
					swap_index = j;
					
				}	
	 	}
		for(i = 0; i < 32; i++)	// swap
			{
				storage[addr*32 + i] = buffer[swap_index*32 + i];
				buffer[swap_index*32 + i] = input[addr*32 + i];
			}
		PAGEFAULT++;
		counter[swap_index]=t;
        pt[swap_index] = addr;
		return swap_index*32+value;
		
	
}


__device__ uchar Gread(uchar *buffer, u32 addr)
{
	u32 frame_num = addr/PAGESIZE;
	u32 offset = addr%PAGESIZE;
	
	addr = paging(buffer, frame_num, offset);
	return buffer[addr];
}


__device__ void Gwrite(uchar *buffer, u32 addr, uchar value)
{
	u32 frame_num = addr/PAGESIZE;
	u32 offset = addr%PAGESIZE;

	addr = paging(buffer , frame_num, offset);
	buffer[addr] = value;
}

__device__ void snapshot(uchar *results, uchar *buffer ,int offset ,int input_size)
{
	for(int i = 0; i < input_size; i++)
		results[i] = Gread(buffer, i+offset);
}


__device__ static void init_pageTable(int pt_entrie){

	for(int i=0;i<pt_entrie;i++){
		pt[i] = -1 ;
	}


}
__global__ void mykernel(int input_size)
{
	__shared__ uchar data[PHYSICAL_MEM_SIZE];
	
	int pt_entries = PHYSICAL_MEM_SIZE/PAGESIZE;
	init_pageTable(pt_entries);
	//####Gwrite/Gread code section start####
	for(int i = 0; i < input_size; i++)
		Gwrite(data,i,input[i]);
	for(int i = input_size - 1 ; i>= input_size - 10 ;i--)
		int value = Gread(data,i);
	snapshot(results,data,0,input_size);
	//####Gwrite/Gread code section end#### 
	printf("pagefault times = %d\n",PAGEFAULT);
	

} 

int load_binaryFile(const char *INPUT_FILE, uchar *input ,int storesize)
{


	FILE *DATA = fopen(INPUT_FILE,"rb");
	int size = 0;
	uchar in;
	while(fread(&in,sizeof(uchar),1,DATA)){
		input[size++] = in;
	}
	fclose(DATA);
	return size;

}


void write_binaryFile(const char *OUT_FILE, uchar *results ,int size){
	FILE *OUT = fopen(OUT_FILE,"wb");
    int i;
	
	for(i=0;i<size;i++){
		fwrite(&results[i],sizeof(uchar),1,OUT);
	}
    fclose(OUT);

}

int main(){
	
	for(int i=0;i<1024;i++)counter[i]=0;
	int input_size = load_binaryFile(DATAFILE, input,STORAGE_SIZE);
	hipSetDevice(0);
	mykernel<<< 1, 1, 16384>>>(input_size);
	hipDeviceSynchronize();
	hipDeviceReset();
	write_binaryFile(OUTFILE,results,input_size);
	return 0;

}
